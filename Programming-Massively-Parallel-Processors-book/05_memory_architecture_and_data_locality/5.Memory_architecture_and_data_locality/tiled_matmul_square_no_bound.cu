#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>

// Basic square Matmul with no boundary checks

#define TILE_WIDTH 16

// Kernel for matrix multiplication using shared memory tiling
__global__ void matrixMulKernel(const float *M, const float *N, float *P,
                                int Width) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  // Block and thread indices
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and column of the P element to work on
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

  float Pvalue = 0.0f;

  // Loop over the M and N tiles required to compute the P element
  for (int ph = 0; ph < Width / TILE_WIDTH; ++ph) {
    // Load the tile from M and N into shared memory
    Mds[ty][tx] = M[Row * Width + (ph * TILE_WIDTH + tx)];
    Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width + Col];

    __syncthreads();

    // Compute partial products
    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }

    __syncthreads();
  }

  // Write the computed value into the result matrix
  P[Row * Width + Col] = Pvalue;
}

int main() {
  // Matrix dimension (must be multiple of TILE_WIDTH to avoid boundary checks),
  // the number of blocks in each dimension (N / TILE_WIDTH) matches the number
  // of tiles needed to cover the matrix.
  int N = 256; // Adjust as desired, but must be divisible by TILE_WIDTH

  // Size in bytes
  size_t size = N * N * sizeof(float);

  // Allocate host memory
  float *h_M = (float *)malloc(size);
  float *h_N = (float *)malloc(size);
  float *h_P = (float *)malloc(size);

  // Initialize host matrices with random values
  srand(0);
  for (int i = 0; i < N * N; i++) {
    h_M[i] = static_cast<float>(rand()) / RAND_MAX;
    h_N[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // Allocate device memory
  float *d_M, *d_N, *d_P;
  hipMalloc((void **)&d_M, size);
  hipMalloc((void **)&d_N, size);
  hipMalloc((void **)&d_P, size);

  // Copy data from host to device
  hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

  // Set up execution configuration
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  dim3 dimGrid(N / TILE_WIDTH, N / TILE_WIDTH);

  // Launch kernel
  matrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, N);

  // Copy result back to host
  hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

  // Optional: Check a few elements for correctness or print a small sub-matrix
  std::cout << "Sample of output matrix (top-left 4x4):" << std::endl;
  for (int i = 0; i < 4; i++) {
    for (int j = 0; j < 4; j++) {
      std::cout << h_P[i * N + j] << "\t";
    }
    std::cout << std::endl;
  }

  // Clean up
  free(h_M);
  free(h_N);
  free(h_P);
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);

  return 0;
}
